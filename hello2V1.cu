/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
  int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
                blockIdx.y * gridDim.x +
               blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z +
                threadIdx.z *  blockDim.x * blockDim.y +
                threadIdx.y * blockDim.x +
                threadIdx.x;

  printf ("Hello Linda Jimenez, 21169 from thread %i\n", myID);

  if (myID == 4096){
    printf("Last message: Hello Linda Jimenez, 21169 from thread %i\n", myID);
  }
}

int main ()
{
  dim3 g(4, 2);    
  dim3 b(32, 16); 
  hello<<<g, b>>>();
  hipDeviceSynchronize();  //use instead, ^ is deprecated
  return 0;
}

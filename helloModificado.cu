#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Linda Jimenez, 21169 from thread %i\n", threadId);

    if (threadId == 1023) {
        printf("Last Message: Linda Jimenez, 21169 from thread %i \n", threadId);
    }
}

int main()
{
    hello<<<1, 2048>>>();
    hipDeviceSynchronize(); //deprecated
    return 0;
}

/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
  int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
                blockIdx.y * gridDim.x +
               blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z +
                threadIdx.z *  blockDim.x * blockDim.y +
                threadIdx.y * blockDim.x +
                threadIdx.x;

  if (myID == 99999){
    printf("Last message: Hello Linda Jimenez, 21169 from thread %i\n", myID);
  }
}

int main ()
{
  dim3 g(391, 1, 1);    
  dim3 b(256, 1, 1); 
  hello<<<g, b>>>();
  hipDeviceSynchronize();  //use instead, ^ is deprecated
  return 0;
}
